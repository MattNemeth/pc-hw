#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include <iostream>
#include <chrono>

#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void blur(unsigned char* in_bm, unsigned char* out_bm) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;
    int maxGridDim = gridDim.x;
    int xmin = -1;
    int xmax = 1;
    int ymin = -1;
    int ymax = 1;
    
    if (x == 0) {
        xmin = 0; 
    } 
    if (x == maxGridDim) { 
        xmax = 0;
    }
    if (y == 0) {
        ymin = 0;
    }
    if (y == maxGridDim) {
        ymax = 0; 
    }

    float val = 0;

    for (int i = ymin; i <= ymax; i++) {
        for (int j = xmin; j <= xmax; j++) {
            int index = x + j + (y + i) * DIM;
	    val += in_bm[index * 4]/9.0;
        }
    }

    out_bm[offset*4] = int(val);
    out_bm[offset*4 + 1] = 0;
    out_bm[offset*4 + 2] = 0;
    out_bm[offset*4 + 3] = 255;
    
}

__global__ void sharpen(unsigned char *in_bm, unsigned char* out_bm) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = (x + y * gridDim.x);
    int xmin = -1;
    int xmax = 1;
    int ymin = -1;
    int ymax = 1;
    int maxGridDim = gridDim.x;
    
    if (x == 0) {
        xmin = 0; 
    } 
    if (x == maxGridDim) { 
        xmax = 0;
    }
    if (y == 0) {
        ymin = 0;
    }
    if (y == maxGridDim) {
        ymax = 0; 
    }

    float val = 0; 

    for (int row = ymin; row <= ymax; row++) {
        for (int col = xmin; col <= xmax; col++) {      
            int index = x + col + (y + row) * maxGridDim;
            double multiplier = 0.0f;
	    if (col == 0 && (row == -1 || row == 1)) {
		multiplier = -0.5f;
	    }
            if (row == 0 && (col == -1 || col == 1)) {
		multiplier = -0.5f;
	    }
            if (row == 0 && col == 0) {
		multiplier = 3.0f;
	    }
            val += in_bm[index * 4] * multiplier;
        }
    }

    out_bm[offset*4] = int(val);
    out_bm[offset*4 + 1] = 0;
    out_bm[offset*4 + 2] = 0;
    out_bm[offset*4 + 3] = 255;
}

__global__ void kernel( unsigned char *ptr ) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    dim3    grid(DIM,DIM);

    DataBlock   orig_data;
    CPUBitmap orig_bitmap( DIM, DIM, &orig_data );
    unsigned char    *orig_dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&orig_dev_bitmap, orig_bitmap.image_size() ) );
    orig_data.dev_bitmap = orig_dev_bitmap;
    kernel<<<grid, 1>>>( orig_dev_bitmap );

    // Setting up Blurred image bitmap
    DataBlock   blur_data;
    CPUBitmap blur_bitmap( DIM, DIM, &blur_data );
    unsigned char    *blur_dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&blur_dev_bitmap, blur_bitmap.image_size() ) );
    blur_data.dev_bitmap = blur_dev_bitmap;
    blur<<<grid, 1>>>( orig_dev_bitmap, blur_dev_bitmap );

    // Setting up Sharpened image bitmap
    DataBlock   sharp_data;
    CPUBitmap sharp_bitmap( DIM, DIM, &sharp_data );
    unsigned char    *sharp_dev_bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&sharp_dev_bitmap, sharp_bitmap.image_size() ) );
    sharp_data.dev_bitmap = sharp_dev_bitmap;
    sharpen<<<grid, 1>>>( orig_dev_bitmap, sharp_dev_bitmap );

    // Handle memory
    HANDLE_ERROR( hipMemcpy( orig_bitmap.get_ptr(), orig_dev_bitmap, orig_bitmap.image_size(), hipMemcpyDeviceToHost ) );    
    HANDLE_ERROR( hipFree( orig_dev_bitmap ) );
    HANDLE_ERROR( hipMemcpy( blur_bitmap.get_ptr(), blur_dev_bitmap, blur_bitmap.image_size(), hipMemcpyDeviceToHost ) );    
    HANDLE_ERROR( hipFree( blur_dev_bitmap ) );
    HANDLE_ERROR( hipMemcpy( sharp_bitmap.get_ptr(), sharp_dev_bitmap, sharp_bitmap.image_size(), hipMemcpyDeviceToHost ) );    
    HANDLE_ERROR( hipFree( sharp_dev_bitmap ) );

    //orig_bitmap.display_and_exit();         
    //blur_bitmap.display_and_exit();         
    sharp_bitmap.display_and_exit();         
}

